#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "opencv2/opencv.hpp"
#include "opencv2/core/core.hpp"
#include "opencv2/core/cuda.hpp"
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <math.h>
//#include "opencv2/cudawarping.hpp"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#define BLUR_SIZE 2

using namespace cv;
using namespace std;

hipError_t blurWithCuda(uchar* in, uchar* out, int w, int h);
//void blurKernel(uchar * in, uchar * out, int w, int h);



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/*
__global__ void blurKernel(uchar* in, uchar* out, int w, int h) {
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Col < w && Row < h) {
        int pixVal = 0;
        int pixels = 0;

        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;

                if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                    pixVal += in[curRow * w + curCol];
                    pixels++;
                }
            }
        }
        out[Row * w + Col] = (unsigned char)(pixVal / pixels);

    }
}
*/

__global__ 
void blurKernel(uchar* in, uchar* out, int w, int h) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < h*BLUR_SIZE && j < w* BLUR_SIZE)
    {
        int iIn = (i / BLUR_SIZE);
        int jIn = (j / BLUR_SIZE);
        out[i * w*BLUR_SIZE + j] = in[iIn * w + jIn];
    }
}

int main(int argc, char* argv[])
{
    Mat image = imread("lena512.bmp", IMREAD_GRAYSCALE);   // Read the file
    //Mat image = imread("lena512.bmp");
    namedWindow("Display window", WINDOW_AUTOSIZE);
    imshow("Display window", image);
    //waitKey(0);


    // import image

    int rows = image.rows;
    int cols = image.cols;

    //float scale = 5.0f;
    uchar* in = image.data;


    uchar * out = (uchar *) malloc(rows * cols * BLUR_SIZE * BLUR_SIZE+1);


    // Add vectors in parallel.

    hipError_t cudaStatus = blurWithCuda(in, out, cols, rows);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    Mat out_mat  = Mat(rows* BLUR_SIZE, cols* BLUR_SIZE, CV_8UC1, out);
    
    namedWindow("Display window2", WINDOW_AUTOSIZE);
    imshow("Display window2", out_mat);
    waitKey(0);
    /*
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    */
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.



hipError_t blurWithCuda(uchar* in, uchar* out, int w, int h)
{
    uchar * dPin;
    uchar * dPout;
    
    hipError_t cudaStatus;

    //dim3 dimGrid(ceil(h / 16.0), ceil(w / 16.0), 1);
    //dim3 dimBlock(16, 16, 1);
    dim3 dimBlock(16, 16);
    dim3 dimGrid( ((w* BLUR_SIZE)/16)+1 , ((h * BLUR_SIZE)/16) +1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dPin, w * h * sizeof(uchar));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dPout, BLUR_SIZE * BLUR_SIZE *w * h * sizeof(uchar));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dPin, in, w * h * sizeof(uchar), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dPout, out, w * h * sizeof(uchar), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
    blurKernel<<< dimGrid, dimBlock >>> (dPin, dPout, w, h);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dPout, w * h * BLUR_SIZE * BLUR_SIZE* sizeof(uchar), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dPin);
    hipFree(dPout);
    
    
    return cudaStatus;
}
